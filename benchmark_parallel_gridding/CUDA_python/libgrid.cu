#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#include "libgrid.h"

__global__ void gridding_kernel(hipDoubleComplex *grid, double *uvwt, hipDoubleComplex *vist, double *freq) {
    int timestep = blockIdx.x;
    int baseline = blockIdx.y;
    int fq = threadIdx.x;

    hipDoubleComplex vis = vist[(timestep * BASELINES * FREQUENCS) + (baseline * FREQUENCS) + fq];
    double f = freq[fq];

    int iu = (int)round(THETA_OVER_C * uvwt[(timestep * BASELINES * 3) + (baseline * 3) + 0] * f);
    int iv = (int)round(THETA_OVER_C * uvwt[(timestep * BASELINES * 3) + (baseline * 3) + 1] * f);
    int iu_idx = iu + IMAGE_SIZE_HALF;
    int iv_idx = iv + IMAGE_SIZE_HALF;

    atomicAdd(&(grid[iu_idx * IMAGE_SIZE + iv_idx].x), hipCreal(vis));
    atomicAdd(&(grid[iu_idx * IMAGE_SIZE + iv_idx].y), hipCimag(vis));
}

void gridding_cuda(std::complex<double> *grid, double *uvwt, std::complex<double> *vist, double *freq) {
    hipDoubleComplex *d_grid;
    double *d_uvwt;
    hipDoubleComplex *d_vist;
    double *d_freq;

    hipMalloc(&d_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    hipMalloc(&d_uvwt, TIMESTEPS * BASELINES * 3 * sizeof(double));
    hipMalloc(&d_vist, TIMESTEPS * BASELINES * FREQUENCS * sizeof(hipDoubleComplex));
    hipMalloc(&d_freq, FREQUENCS * sizeof(double));

    // no need to copy the grid since it's zero initialized on the host
    hipMemset(d_grid, 0, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex));
    hipMemcpy(d_uvwt, uvwt, TIMESTEPS * BASELINES * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_vist, vist, TIMESTEPS * BASELINES * FREQUENCS * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_freq, freq, FREQUENCS * sizeof(double), hipMemcpyHostToDevice);

    dim3 gridDim(TIMESTEPS, BASELINES);
    dim3 blockDim(FREQUENCS);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    gridding_kernel<<<gridDim, blockDim>>>(d_grid, d_uvwt, d_vist, d_freq);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("kernel execution time: %f ms\n", milliseconds);

    hipMemcpy(grid, d_grid, IMAGE_SIZE * IMAGE_SIZE * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    hipFree(d_grid);
    hipFree(d_uvwt);
    hipFree(d_vist);
    hipFree(d_freq);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

